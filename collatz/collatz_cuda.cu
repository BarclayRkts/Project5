/*
Collatz code

Copyright 2022 Martin Burtscher

Redistribution in source or binary form, with or without modification, is not
permitted. Use in source or binary form, with or without modification, is only
permitted for academic use in CS 4380 and CS 5351 at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/


#include <cstdio>
#include <algorithm>
#include <iostream>
#include <sys/time.h>
#include <hip/hip_runtime.h>

static const int ThreadsPerBlock = 512;

static __global__ void collatz(const long start, const long top, int* maxlen){
  //int maxlen = 0;
  //printf("here1");
  // compute sequence lengths
  //for (long i = start; i < top; i += 2) {
    const long idx = threadIdx.x * blockIdx.x * blockDim.x;
    long val = idx;
    int len = 1;
    do {
      //std::cout << "Here2" << std::endl;
      len++;
      if ((val % 2) != 0) {
        val = 3 * val + 1;  // odd
      } else {
        val = val / 2;  // even
      }
    } while (val != 1);

    atomicMax(maxlen, len);
    //maxlen = std::max(*maxlen, len);
  //}

  //return maxlen;
}

static void CheckCuda(const int line)
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "CUDA error %d on line %d: %s\n", e, line, hipGetErrorString(e));
    exit(-1);
  }
}

int main(int argc, char* argv [])
{
  printf("Collatz v1.8\n");

  // check command line
  if (argc != 3) {fprintf(stderr, "USAGE: %s start_value upper_bound\n", argv[0]); exit(-1);}
  const long start = atol(argv[1]);
  const long top = atol(argv[2]);
  printf("start value: %ld\n", start);
  printf("upper bound: %ld\n", top);
  
  // allocate vectors on GPU
  int maxlen = 0;
  int* maxlen_gpu;
  hipMalloc((void **)&maxlen_gpu, sizeof(int));
  CheckCuda(__LINE__);

  // initialize vectors on GPU
  hipMemcpy(maxlen_gpu, &maxlen, sizeof(int), hipMemcpyHostToDevice);
  CheckCuda(__LINE__);

  // start time
  timeval beg, end;
  gettimeofday(&beg, NULL);
  CheckCuda(__LINE__);

  // execute timed code
  //const int maxlen = collatz(start, top);
 int block = ((((top-start)+1)/2) + ThreadsPerBlock - 1);
 std::cout << "number of blocks: " << block / ThreadsPerBlock << std::endl;
  //printf("block: ", (maxlen + ThreadsPerBlock - 1) / ThreadsPerBlock);
  collatz<<< block / ThreadsPerBlock, ThreadsPerBlock>>>(start, top, maxlen_gpu);
  
  hipDeviceSynchronize();

  // end time
  gettimeofday(&end, NULL);
  const double runtime = end.tv_sec - beg.tv_sec + (end.tv_usec - beg.tv_usec) / 1000000.0;
  printf("compute time: %.6f s\n", runtime);

    // get result from GPU
  hipMemcpy(&maxlen, maxlen_gpu, sizeof(int), hipMemcpyDeviceToHost);
  CheckCuda(__LINE__);

  // print result
  printf("maximum length: %d\n", maxlen);
  return 0;
}
